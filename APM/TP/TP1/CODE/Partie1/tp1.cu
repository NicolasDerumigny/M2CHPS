
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void kernel(double *a, double *b, double *c, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    
    if (i < N)
    {
	c[i] = a[i] + b[i];
    }
}

int main(int argc, char **argv)
{
    int N = 1000;
    int sz_in_bytes = N*sizeof(double);

    double *h_a, *h_b, *h_c;
    double *d_a, *d_b, *d_c;

    h_a = (double*)malloc(sz_in_bytes);
    h_b = (double*)malloc(sz_in_bytes);
    h_c = (double*)malloc(sz_in_bytes);

    // Initiate values on h_a and h_b
    for(int i = 0 ; i < N ; i++)
    {
	h_a[i] = 1./(1.+i);
	h_b[i] = (i-1.)/(i+1.);
    }

    // 3-arrays allocation on device 
    hipMalloc((void**)&d_a, sz_in_bytes);
    hipMalloc((void**)&d_b, sz_in_bytes);
    hipMalloc((void**)&d_c, sz_in_bytes);

    // copy on device values pointed on host by h_a and h_b
    // (the new values are pointed by d_a et d_b on device)
    hipMemcpy(d_a, h_a, sz_in_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sz_in_bytes, hipMemcpyHostToDevice);

    dim3  dimBlock(64, 1, 1);
    dim3  dimGrid((N + dimBlock.x - 1)/dimBlock.x, 1, 1);
    kernel<<<dimGrid , dimBlock>>>(d_a, d_b, d_c, N);

    // Result is pointed by d_c on device
    // Copy this result on host (result pointed by h_c on host)
    hipMemcpy(h_c, d_c, sz_in_bytes, hipMemcpyDeviceToHost);

    // freeing on device 
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
