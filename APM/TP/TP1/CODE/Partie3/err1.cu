
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void kernel(double *a, double *b, double *c, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    c[i] = a[i] + b[i];
}

int main(int argc, char **argv)
{
    int N = 1000;
    int sz_in_bytes = N*sizeof(double);

    double *h_a, *h_b, *h_c;
    double *d_a, *d_b, *d_c;

    h_a = (double*)malloc(sz_in_bytes);
    h_b = (double*)malloc(sz_in_bytes);
    h_c = (double*)malloc(sz_in_bytes);

    // Initiate values on h_a and h_b
    for(int i = 0 ; i < N ; i++)
    {
	h_a[i] = 1./(1.+i);
	h_b[i] = (i-1.)/(i+1.);
    }

    hipMalloc((void**)&d_a, sz_in_bytes);
    hipMalloc((void**)&d_b, 0);
    hipMalloc((void**)&d_c, sz_in_bytes);

    hipMemcpy(d_a, h_a, sz_in_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sz_in_bytes, hipMemcpyHostToDevice);

    dim3  dimBlock(64, 1, 1);
    dim3  dimGrid(10, 1, 1);
    kernel<<<dimGrid , dimBlock>>>(d_a, d_b, d_c, N);

    hipMemcpy(h_c, d_c, sz_in_bytes, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Verifying
    double err = 0, norm = 0;
    for(int i = 0 ; i < N ; i++)
    {
	double err_loc = fabs(h_c[i] - (h_a[i]+h_b[i]));
	err  += err_loc;
	norm += fabs(h_c[i]);
    }
    if (err/norm < 1.e-16)
    {
	printf("SUCCESS (Relative error : %.3e)\n", err/norm);
    }
    else
    {
	printf("ERROR (Relative error : %.3e)\n", err/norm);
    }

    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}

