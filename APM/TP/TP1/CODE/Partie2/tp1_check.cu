
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void kernel(double *a, double *b, double *c, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    
    if (i < N)
    {
	c[i] = a[i] + b[i];
    }
}

int main(int argc, char **argv)
{
				int N = 1000;
				int sz_in_bytes = N*sizeof(double);

				double *h_a, *h_b, *h_c;
				double *d_a, *d_b, *d_c;

				h_a = (double*)malloc(sz_in_bytes);
				h_b = (double*)malloc(sz_in_bytes);
				h_c = (double*)malloc(sz_in_bytes);

				// Initiate values on h_a and h_b
				for(int i = 0 ; i < N ; i++)
				{
								h_a[i] = 1./(1.+i);
								h_b[i] = (i-1.)/(i+1.);
				}

				// emulate allocation on device (similar to cudaMalloc)
				hipMalloc((void**)&d_a, sz_in_bytes);
				hipMalloc((void**)&d_b, sz_in_bytes);
				hipMalloc((void**)&d_c, sz_in_bytes);

				// emulate h_a and h_b arrays on device
				hipMemcpy(d_a, h_a, sz_in_bytes, hipMemcpyHostToDevice);
				hipMemcpy(d_b, h_b, sz_in_bytes, hipMemcpyHostToDevice);

				dim3  dimBlock(64, 1, 1);
				dim3  dimGrid((N + dimBlock.x - 1)/dimBlock.x, 1, 1);
				kernel<<<dimGrid , dimBlock>>>(d_a, d_b, d_c, N);

				hipMemcpy(h_c, d_c, sz_in_bytes, hipMemcpyDeviceToHost);

				// emulate freeing on device (similar to cudaFree)
				hipFree(d_a);
				hipFree(d_b);
				hipFree(d_c);

				// Verifying
				double err = 0, norm = 0;
				for(int i = 0 ; i < N ; i++)
				{
								double err_loc = fabs(h_c[i] - (h_a[i]+h_b[i]));
								err  += err_loc;
								norm += fabs(h_c[i]);
				}
				printf("Relative error : %.3e\n", err/norm);

				free(h_a);
				free(h_b);
				free(h_c);

				return 0;
}
