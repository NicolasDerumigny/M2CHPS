#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define SIZE 102400
#define MOD 102399
#define STEP 128

/* ARRAY A INITIALIZER */
void init_a(int * a)
{
    int i;
    for(i=0; i<SIZE; i++)
    {
        a[i] = 1;
    }
}

/* ARRAY B INITIALIZER */
void init_b(int * b)
{
	int i, j;

	j=0;

	for(i=0; i<SIZE-1; i++)
	{
		b[j] = i;
		j = (j+STEP)%MOD;
	}	

    b[SIZE-1] = SIZE-1;
}

/* CHECKING A VALUES */
int check_a(int * a)
{
    int i;
    int correct = 1;
	for(i=0; i<SIZE; i++)
	{
		if(a[i] != (i+1)) 
		{
         
			correct = 0;
		} 
	}	

    return correct;
}


/* CUDA FUNCTION */
__global__ void mykernel(int * a, int * b, int N)
{
/* A COMPLETER */
}


int main(int argc, char * argv[])
{

	int * a = (int *)malloc(sizeof(int)*SIZE);
	int * b = (int *)malloc(sizeof(int)*SIZE);

    init_a(a);
	init_b(b);


/*  INSERT CUDA ALLOCATION AND COPY HERE */
    /* A COMPLETER */    


	dim3 nBlocks;
	dim3 nThperBlock;

	nBlocks.x = 1;
	
	nThperBlock.x = 1024;

	mykernel<<< nBlocks , nThperBlock >>>(d_a, d_b, SIZE);

	
/* INSERT CUDA COPY HERE */
    /* A COMPLETER */

	int correct = check_a(a);;
	
	if(0 == correct)
	{
		printf("\n\n ******************** \n ***/!\\ ERROR /!\\ *** \n ******************** \n\n");
	}
	else
	{
		printf("\n\n ******************** \n ***** SUCCESS! ***** \n ******************** \n\n");
	}


	return 1;
}
