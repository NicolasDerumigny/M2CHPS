#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>


#define SIZE 10240
#define NBBLOCKS 1


/* B INITIALIZER */
void init_b(int * b)
{
	int i, j;
	int * check = malloc(sizeof(int)*SIZE);
	memset(b, -1, SIZE*sizeof(int));
	memset(check, 0, SIZE*sizeof(int));

	for(i=0; i<SIZE*2; i++)
	{
		int value = (i*i)%SIZE;
		if(0 == check[value])
		{	
			b[i] = value;
			check[value]=1;
		}
	}	
	for(i=0; i<SIZE; i++)
	{
		if(0 == check[i])
		{
		for(j=0; j<SIZE && (0 == check[i]); j++)
			{
				if(-1 == b[j])
				{ 
					b[j] = i;
					check[i] = 1;
				}
			}
		}
	}
	free(check);
}




/* CUDA FUNCTION */
__global__ mykernel(int * a, int * b, int N)
{
//	int i;
//	for(i=0; i<SIZE; i++)
//	{
//		a[b[i]] = i;
//	}

}


int main(int argc, char * argv[])
{

	int * a = malloc(sizeof(int)*SIZE);
	int * b = malloc(sizeof(int)*SIZE);

	init_b(b);



/*  INSERT CUDA ALLOCATION AND COPY HERE */
	

/* FILL IN MYKERNEL ARGS*/

	dim3 nBlocks;
	dim3 nThperBlock;

	nBlock.x = 1;
	
	nThperBlock = SIZE / 1;

	mykernel<<< nBlock , nThperBlock >>>(d_a, d_b, SIZE);

	
/* INSERT CUDA COPY HERE */





	int correct = 1;
	
	for(i=0; i<SIZE; i++)
	{
		if(a[i] != i) 
		{
			correct = 0;
		} 
	}	


	if(0 == correct)
	{
		printf("ERROR!!!!! \n");
	}
	else
	{
		printf("SUCCESS! \n");
	}


	return 1;
}
