#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "libvideo.h"

#define SEUIL 50

__global__ void kernel_sobel(char * frame, int width, int length)
{
/* A COMPLETER */
}



int main (int argc, char * argv[])
{
	int i, j, k;
	int cpt_frame;
	int cpt = 0;
	int frame_count;
	int width, height;



	printf("Opening videos - read and write\n"); fflush(stdout);

	OpenReadAndWriteVideo("./Wildlife.wmv", "./my_copy.wmv");


	printf("----------------------------------------\n");
	frame_count = getFrameCount();
	width = getWidth();
	height = getHeight();
	printf("Frame count = %d\n", frame_count); fflush(stdout);

	printf("Width  of frames: %d\n", width); fflush(stdout);
	printf("Height of frames: %d\n", height); fflush(stdout);


//	char * frames = (char *) malloc( sizeof(char) * frame_count * width * height * 3);
	char * frame1 = (char *) malloc( sizeof(char) * width * height * 3);

	char * sobel;
	sobel = (char *)malloc(sizeof(char) * width*height*3);

	int line_size = width*3;

	for(cpt_frame = 0; cpt_frame < 500 && cpt_frame < frame_count; cpt_frame ++)
	{

		printf("%d - Read frame with index\n", cpt_frame); fflush(stdout);
		readFrame_with_index(frame1, cpt_frame);


		if(cpt_frame > 200 && cpt_frame < 400)
		{
			printf("%d - GREY\n", cpt_frame); fflush(stdout);


			for(i=0; i<height; i++)
			{
				for(j=0; j<width; j++)
				{
					//					pixel = frame->at<Vec3b>(i, j);	

					int red   = frame1[i*line_size+3*j+0];
					int green = frame1[i*line_size+3*j+1];
					int blue  = frame1[i*line_size+3*j+2];

					int moy = red/4 + (green*3/4);

					frame1[i*line_size+3*j+0] = (char)moy;
					frame1[i*line_size+3*j+1] = (char)moy;
					frame1[i*line_size+3*j+2] = (char)moy;


				}
			}

		}
		if(cpt_frame >= 400 && cpt_frame < 800)
		{
			printf("%d - SOBEL\n", cpt_frame); fflush(stdout); 


			/***************************************/
			/****************** SOBEL **************/
			/***************************************/

            /* A COMPLETER */


		}


//		printf("%d - Write frame\n", cpt); fflush(stdout);
		writeFrame (frame1);
//		printf("%d - Frame written\n", cpt); fflush(stdout);

	}
	printf("ECRITURE VIDEO FINIE\n");

	free(frame1);
	free(sobel);



	return 0;

}
