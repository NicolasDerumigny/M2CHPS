#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "libvideo.h"

#define SEUIL 50

__global__ void kernel_grey(char * frame, int width, int length)
{
/* A COMPLETER */
}



int main (int argc, char * argv[])
{
	int i, j, k;
	int cpt_frame;
	int cpt = 0;
	int frame_count;
	int width, height;



	printf("Opening videos - read and write\n"); fflush(stdout);

	OpenReadAndWriteVideo("./Wildlife.wmv", "./my_copy2.wmv");


	printf("----------------------------------------\n");
	frame_count = getFrameCount();
	width = getWidth();
	height = getHeight();
	printf("Frame count = %d\n", frame_count); fflush(stdout);

	printf("Width  of frames: %d\n", width); fflush(stdout);
	printf("Height of frames: %d\n", height); fflush(stdout);


//	char * frames = (char *) malloc( sizeof(char) * frame_count * width * height * 3);
	char * frame1 = (char *) malloc( sizeof(char) * width * height * 3);

	int line_size = width*3;

	for(cpt_frame = 0; cpt_frame < 500 && cpt_frame < frame_count; cpt_frame ++)
	{

		printf("%d - Read frame with index\n", cpt_frame); fflush(stdout);
		readFrame_with_index(frame1, cpt_frame);


		if(cpt_frame > 200 && cpt_frame < 400)
		{
			printf("%d - GREY\n", cpt_frame); fflush(stdout);

        /* COLOR -> GREY */
        /* A COMPLETER */


		}
		writeFrame (frame1);

	}
	printf("ECRITURE VIDEO FINIE\n");

	free(frame1);
	free(sobel);



	return 0;

}
